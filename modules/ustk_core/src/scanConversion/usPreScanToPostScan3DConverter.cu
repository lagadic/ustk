#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * This file is part of the ustk software.
 * Copyright (C) 2016 - 2017 by Inria. All rights reserved.
 *
 * This software is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * ("GPL") version 2 as published by the Free Software Foundation.
 * See the file LICENSE.txt at the root directory of this source
 * distribution for additional information about the GNU GPL.
 *
 * For using ustk with software that can not be combined with the GNU
 * GPL, please contact Inria about acquiring a ViSP Professional
 * Edition License.
 *
 * This software was developed at:
 * Inria Rennes - Bretagne Atlantique
 * Campus Universitaire de Beaulieu
 * 35042 Rennes Cedex
 * France
 *
 * If you have questions regarding the use of this file, please contact
 * Inria at ustk@inria.fr
 *
 * This file is provided AS IS with NO WARRANTY OF ANY KIND, INCLUDING THE
 * WARRANTY OF DESIGN, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
 *
 * Authors:
 * Jason Chevrie
 *
 *****************************************************************************/


#include <visp3/ustk_core/usConfig.h>

#if defined(USTK_HAVE_CUDA)
#include <hip/hip_runtime.h>

#include <visp3/core/vpException.h>
#include <visp3/ustk_core/usPreScanToPostScan3DConverter.h>

struct cudaVoxelWeightAndIndex
{
  unsigned int m_outputIndex;
  unsigned int m_inputIndex[8];
  double m_W[8];
};

struct cudaVoxelWeightAndIndexReducedMemory
{
  unsigned int m_outputIndex;
  unsigned int m_inputIndex;
  double m_W[3];
};

__global__ void kernelPostScanVoxelDirect(unsigned char *dataPost, const unsigned char *dataPre, unsigned int m_nbX, unsigned int m_nbY, unsigned int m_nbZ, int X, int Y, int Z, double m_resolution, double xmax, double ymin, double zmax, unsigned int frameNumber, unsigned int scanLineNumber, double transducerRadius, double motorRadius, double scanLinePitch, double axialResolution, double framePitch, bool sweepInZdirection)
{
  uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
  uint z = (blockIdx.z * blockDim.z) + threadIdx.z;

  if (x >= m_nbX  || y >= m_nbY || z >= m_nbZ) return;

  unsigned int nbXY = m_nbX * m_nbY;
  unsigned int XY = X * Y;

  double xx = m_resolution * x - xmax;
  double yy = ymin + m_resolution * y;
  double zz = m_resolution * z - zmax;

  double i, j, k;

  double radiusOffset = transducerRadius - motorRadius;
  double rProbe = radiusOffset + sqrt(yy * yy + zz * zz);
  double r = sqrt(rProbe * rProbe + xx * xx);
  double phi = atan(xx / rProbe);
  double theta = atan(zz / yy);

  double itmp = phi / scanLinePitch + 0.5 * (scanLineNumber - 1);
  i = itmp;
  j = (r - transducerRadius) / axialResolution;
  k = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (sweepInZdirection ? itmp : scanLineNumber-1-itmp) / scanLineNumber;

  double ii = floor(i);
  double jj = floor(j);
  double kk = floor(k);

  if (ii >= 0 && jj >= 0 && kk >= 0 && ii + 1 < X && jj + 1 < Y && kk + 1 < Z) {

    double u = i - ii;
    double v = j - jj;
    double w = k - kk;
    double u1 = 1 - u;
    double v1 = 1 - v;
    double w1 = 1 - w;

    double v1w1 = v1 * w1;
    double vw1 = v * w1;
    double v1w = v1 * w;
    double vw = v * w;

    double W[8] = { u1 * v1w1,
                    u * v1w1,
                    u1 * vw1,
                    u * vw1,
                    u1 * v1w,
                    u * v1w,
                    u1 * vw,
                    u * vw };

    double Xjj = X * jj;
    double Xjj1 = X * (jj + 1);
    double XYKK = XY * kk;
    double XYKK1 = XY * (kk + 1);

    unsigned int index[8] = { (unsigned int)(ii + Xjj + XYKK),
                              (unsigned int)(ii + 1 + Xjj + XYKK),
                              (unsigned int)(ii + Xjj1 + XYKK),
                              (unsigned int)(ii + 1 + Xjj1 + XYKK),
                              (unsigned int)(ii + Xjj + XYKK1),
                              (unsigned int)(ii + 1 + Xjj + XYKK1),
                              (unsigned int)(ii + Xjj1 + XYKK1),
                              (unsigned int)(ii + 1 + Xjj1 + XYKK1) };

    double val = 0;
    for (int n = 0; n < 8; n++) val += W[n] * dataPre[index[n]];
    dataPost[x + m_nbX * y + nbXY * z] = (unsigned char)val;
  }
}

__global__ void kernelPostScanVoxelFillFullLookUpTable(cudaVoxelWeightAndIndex *lookupTable1, cudaVoxelWeightAndIndex *lookupTable2, unsigned int m_nbX, unsigned int m_nbY, unsigned int m_nbZ, int X, int Y, int Z, double m_resolution, double xmax, double ymin, double zmax, unsigned int frameNumber, unsigned int scanLineNumber, double transducerRadius, double motorRadius, double scanLinePitch, double axialResolution, double framePitch)
{
  uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
  uint z = (blockIdx.z * blockDim.z) + threadIdx.z;

  if (x >= m_nbX  || y >= m_nbY || z >= m_nbZ) return;

  unsigned int nbXY = m_nbX * m_nbY;
  unsigned int XY = X * Y;

  double xx = m_resolution * x - xmax;
  double yy = ymin + m_resolution * y;
  double zz = m_resolution * z - zmax;

  double i, j, k1, k2;

  double radiusOffset = transducerRadius - motorRadius;
  double rProbe = radiusOffset + sqrt(yy * yy + zz * zz);
  double r = sqrt(rProbe * rProbe + xx * xx);
  double phi = atan(xx / rProbe);
  double theta = atan(zz / yy);

  double itmp = phi / scanLinePitch + 0.5 * (scanLineNumber - 1);
  i = itmp;
  j = (r - transducerRadius) / axialResolution;

  k1 = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (itmp) / scanLineNumber;

  k2 = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (scanLineNumber-1-itmp) / scanLineNumber;

  double ii = floor(i);
  double jj = floor(j);
  double kk1 = floor(k1);
  double kk2 = floor(k2);

  if (ii >= 0 && jj >= 0 && kk1 >= 0 && ii + 1 < X && jj + 1 < Y && kk1 + 1 < Z) {
    cudaVoxelWeightAndIndex m;

    m.m_outputIndex = x + m_nbX * y + nbXY * z;

    double u = i - ii;
    double v = j - jj;
    double w = k1 - kk1;
    double u1 = 1 - u;
    double v1 = 1 - v;
    double w1 = 1 - w;

    double v1w1 = v1 * w1;
    double vw1 = v * w1;
    double v1w = v1 * w;
    double vw = v * w;

    m.m_W[0] = u1 * v1w1;
    m.m_W[1] = u * v1w1;
    m.m_W[2] = u1 * vw1;
    m.m_W[3] = u * vw1;
    m.m_W[4] = u1 * v1w;
    m.m_W[5] = u * v1w;
    m.m_W[6] = u1 * vw;
    m.m_W[7] = u * vw;

    double Xjj = X * jj;
    double Xjj1 = X * (jj + 1);
    double XYKK = XY * kk1;
    double XYKK1 = XY * (kk1 + 1);

    m.m_inputIndex[0] = (unsigned int)(ii + Xjj + XYKK);
    m.m_inputIndex[1] = (unsigned int)(ii + 1 + Xjj + XYKK);
    m.m_inputIndex[2] = (unsigned int)(ii + Xjj1 + XYKK);
    m.m_inputIndex[3] = (unsigned int)(ii + 1 + Xjj1 + XYKK);
    m.m_inputIndex[4] = (unsigned int)(ii + Xjj + XYKK1);
    m.m_inputIndex[5] = (unsigned int)(ii + 1 + Xjj + XYKK1);
    m.m_inputIndex[6] = (unsigned int)(ii + Xjj1 + XYKK1);
    m.m_inputIndex[7] = (unsigned int)(ii + 1 + Xjj1 + XYKK1);

    lookupTable1[m.m_outputIndex] = m;
  }
  if (ii >= 0 && jj >= 0 && kk2 >= 0 && ii + 1 < X && jj + 1 < Y && kk2 + 1 < Z) {
    cudaVoxelWeightAndIndex m;

    m.m_outputIndex = x + m_nbX * y + nbXY * z;

    double u = i - ii;
    double v = j - jj;
    double w = k2 - kk2;
    double u1 = 1 - u;
    double v1 = 1 - v;
    double w1 = 1 - w;

    double v1w1 = v1 * w1;
    double vw1 = v * w1;
    double v1w = v1 * w;
    double vw = v * w;

    m.m_W[0] = u1 * v1w1;
    m.m_W[1] = u * v1w1;
    m.m_W[2] = u1 * vw1;
    m.m_W[3] = u * vw1;
    m.m_W[4] = u1 * v1w;
    m.m_W[5] = u * v1w;
    m.m_W[6] = u1 * vw;
    m.m_W[7] = u * vw;

    double Xjj = X * jj;
    double Xjj1 = X * (jj + 1);
    double XYKK = XY * kk2;
    double XYKK1 = XY * (kk2 + 1);

    m.m_inputIndex[0] = (unsigned int)(ii + Xjj + XYKK);
    m.m_inputIndex[1] = (unsigned int)(ii + 1 + Xjj + XYKK);
    m.m_inputIndex[2] = (unsigned int)(ii + Xjj1 + XYKK);
    m.m_inputIndex[3] = (unsigned int)(ii + 1 + Xjj1 + XYKK);
    m.m_inputIndex[4] = (unsigned int)(ii + Xjj + XYKK1);
    m.m_inputIndex[5] = (unsigned int)(ii + 1 + Xjj + XYKK1);
    m.m_inputIndex[6] = (unsigned int)(ii + Xjj1 + XYKK1);
    m.m_inputIndex[7] = (unsigned int)(ii + 1 + Xjj1 + XYKK1);

    lookupTable2[m.m_outputIndex] = m;
  }
}

__global__ void kernelPostScanVoxelFullLookUpTable(unsigned char *dataPost, const unsigned char *dataPre, const cudaVoxelWeightAndIndex *lookupTable, long int size)
{
  uint index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index >= size) return;

  double v = 0;
  for (int j = 0; j < 8; j++)
    v += lookupTable[index].m_W[j] * dataPre[lookupTable[index].m_inputIndex[j]];
  dataPost[lookupTable[index].m_outputIndex] = (unsigned char)v;
}

__global__ void kernelPostScanVoxelFillReducedLookUpTable(cudaVoxelWeightAndIndexReducedMemory *lookupTable1, cudaVoxelWeightAndIndexReducedMemory *lookupTable2, unsigned int m_nbX, unsigned int m_nbY, unsigned int m_nbZ, int X, int Y, int Z, double m_resolution, double xmax, double ymin, double zmax, unsigned int frameNumber, unsigned int scanLineNumber, double transducerRadius, double motorRadius, double scanLinePitch, double axialResolution, double framePitch)
{
  uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
  uint z = (blockIdx.z * blockDim.z) + threadIdx.z;

  if (x >= m_nbX  || y >= m_nbY || z >= m_nbZ) return;

  unsigned int nbXY = m_nbX * m_nbY;
  unsigned int XY = X * Y;

  double xx = m_resolution * x - xmax;
  double yy = ymin + m_resolution * y;
  double zz = m_resolution * z - zmax;

  double i, j, k1, k2;

  double radiusOffset = transducerRadius - motorRadius;
  double rProbe = radiusOffset + sqrt(yy * yy + zz * zz);
  double r = sqrt(rProbe * rProbe + xx * xx);
  double phi = atan(xx / rProbe);
  double theta = atan(zz / yy);

  double itmp = phi / scanLinePitch + 0.5 * (scanLineNumber - 1);
  i = itmp;
  j = (r - transducerRadius) / axialResolution;

  k1 = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (itmp) / scanLineNumber;

  k2 = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (scanLineNumber-1-itmp) / scanLineNumber;

  double ii = floor(i);
  double jj = floor(j);
  double kk1 = floor(k1);
  double kk2 = floor(k2);

  if (ii >= 0 && jj >= 0 && kk1 >= 0 && ii + 1 < X && jj + 1 < Y && kk1 + 1 < Z) {
    cudaVoxelWeightAndIndexReducedMemory m;

    m.m_outputIndex = x + m_nbX * y + nbXY * z;

    m.m_W[0] = i - ii;
    m.m_W[1] = j - jj;
    m.m_W[2] = k1 - kk1;

    m.m_inputIndex = (unsigned int)(ii + X * jj + XY * kk1);

    lookupTable1[m.m_outputIndex] = m;
  }
  if (ii >= 0 && jj >= 0 && kk2 >= 0 && ii + 1 < X && jj + 1 < Y && kk2 + 1 < Z) {
    cudaVoxelWeightAndIndexReducedMemory m;

    m.m_outputIndex = x + m_nbX * y + nbXY * z;

    m.m_W[0] = i - ii;
    m.m_W[1] = j - jj;
    m.m_W[2] = k2 - kk2;

    m.m_inputIndex = (unsigned int)(ii + X * jj + XY * kk2);

    lookupTable2[m.m_outputIndex] = m;
  }
}

__global__ void kernelPostScanVoxelReducedLookUpTable(unsigned char *dataPost, const unsigned char *dataPre, const cudaVoxelWeightAndIndexReducedMemory *lookupTable, long int size, int X, int Y)
{
  uint index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index >= size) return;

  const cudaVoxelWeightAndIndexReducedMemory &m = lookupTable[index];
  double u = m.m_W[0];
  double v = m.m_W[1];
  double w = m.m_W[2];
  double u1 = 1 - u;
  double v1 = 1 - v;
  double w1 = 1 - w;

  double v1w1 = v1 * w1;
  double vw1 = v * w1;
  double v1w = v1 * w;
  double vw = v * w;

  double W[8] = { u1 * v1w1, u * v1w1, u1 * vw1, u * vw1, u1 * v1w, u * v1w, u1 * vw, u * vw };

  unsigned int XY = X*Y;
  unsigned int inputIndex[8] = { m.m_inputIndex,          m.m_inputIndex + 1,         m.m_inputIndex + X,
                             m.m_inputIndex + 1 + X,  m.m_inputIndex + XY,        m.m_inputIndex + 1 + XY,
                             m.m_inputIndex + X + XY, m.m_inputIndex + 1 + X + XY };

  double val = 0;
  for (int j = 0; j < 8; j++)
    val += W[j] * dataPre[inputIndex[j]];
  dataPost[m.m_outputIndex] = (unsigned char)val;
}

void usPreScanToPostScan3DConverter::GPUDirectConversion(unsigned char *dataPost, const unsigned char *dataPre)
{
  int X = m_VpreScan.getWidth();
  int Y = m_VpreScan.getHeight();
  int Z = m_VpreScan.getNumberOfFrames();

  double xmax;
  double ymin;
  double ymax;
  double zmax;

  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord(0.0, X, Z, &ymin, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z / 2.0, &ymax, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, (double)X, Z / 2.0, NULL, &xmax, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z, NULL, NULL, &zmax);

  unsigned char *dataPostDevice;
  unsigned int sizePost = m_nbX*m_nbY*m_nbZ*sizeof(unsigned char);
  unsigned char *dataPreDevice;
  unsigned int sizePre = X*Y*Z*sizeof(unsigned char);

  hipError_t codePost = hipMalloc((void **)&dataPostDevice, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversion: GPU post-scan memory allocation error (%d Bytes): %s", sizePost, hipGetErrorString(codePost));
  hipError_t codePre = hipMalloc((void **)&dataPreDevice, sizePre);
  if (codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversion: GPU pre-scan memory allocation error (%d Bytes): %s", sizePre, hipGetErrorString(codePre));

  codePost = hipMemset(dataPostDevice, 0, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU post-scan memory set error: %s", hipGetErrorString(codePost));
  codePre = hipMemcpy(dataPreDevice, dataPre, sizePre, hipMemcpyHostToDevice);
  if (codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU pre-scan memory copy error: %s", hipGetErrorString(codePre));

  dim3 threadsPerBlock(8, 8, 8);
  dim3 numBlocks((m_nbX+threadsPerBlock.x-1)/threadsPerBlock.x, (m_nbY+threadsPerBlock.y-1)/threadsPerBlock.y, (m_nbZ+threadsPerBlock.z-1)/threadsPerBlock.z);
  kernelPostScanVoxelDirect<<<numBlocks, threadsPerBlock>>>(dataPostDevice, dataPreDevice, m_nbX, m_nbY, m_nbZ, X, Y, Z, m_resolution, xmax, ymin, zmax, m_VpreScan.getFrameNumber(), m_VpreScan.getScanLineNumber(), m_VpreScan.getTransducerRadius(), m_VpreScan.getMotorRadius(), m_VpreScan.getScanLinePitch(), m_VpreScan.getAxialResolution(), m_VpreScan.getFramePitch(), m_SweepInZdirection);

  codePost = hipMemcpy(dataPost, dataPostDevice, sizePost, hipMemcpyDeviceToHost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU post-scan memory copy error: %s", hipGetErrorString(codePost));
  hipFree(dataPostDevice);
  hipFree(dataPreDevice);
}

void usPreScanToPostScan3DConverter::GPUFreeLookupTables()
{
  if (m_GPULookupTables[0] != NULL) hipFree(m_GPULookupTables[0]);
  m_GPULookupTablesSize[0] = 0;
  if (m_GPULookupTables[1] != NULL) hipFree(m_GPULookupTables[1]);
  m_GPULookupTablesSize[1] = 0;
}

void usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables()
{
  if (m_GPULookupTables[0] != NULL || m_GPULookupTables[1] != NULL) this->GPUFreeLookupTables();

  m_GPULookupTablesSize[0] = (long int)m_nbX * (long int)m_nbY * (long int)m_nbZ;
  m_GPULookupTablesSize[1] = m_GPULookupTablesSize[0];
  long int LUTmaxSize = m_GPULookupTablesSize[0] * sizeof(cudaVoxelWeightAndIndex);

  hipError_t code = hipMalloc((void **)&(m_GPULookupTables[0]), LUTmaxSize);
  if (code != hipSuccess) {
    std::cout << "Warning: usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: GPU memory allocation error for table 1 (" << LUTmaxSize << " Bytes): " << hipGetErrorString(code) << "\nwill try on RAM, performances can be impacted" << std::endl;
    code = hipMallocManaged((void **)&(m_GPULookupTables[0]), LUTmaxSize);
    if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: memory allocation error for table 1 (%d Bytes): %s", LUTmaxSize, hipGetErrorString(code));
  }

  code = hipMalloc((void **)&(m_GPULookupTables[1]), LUTmaxSize);
  if (code != hipSuccess) {
    std::cout << "Warning: usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: GPU memory allocation error for table 2 (" << LUTmaxSize << " Bytes): " << hipGetErrorString(code) << "\nwill try on RAM, performances can be impacted" << std::endl;
    code = hipMallocManaged((void **)&(m_GPULookupTables[1]), LUTmaxSize);
    if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: memory allocation error for table 2 (%d Bytes): %s", LUTmaxSize, hipGetErrorString(code));
  }

  code = hipMemset(m_GPULookupTables[0], 0, LUTmaxSize);
  if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: GPU memory set error for table 1: %s", hipGetErrorString(code));
  code = hipMemset(m_GPULookupTables[1], 0, LUTmaxSize);
  if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateFullLookupTables: GPU memory set error for table 2: %s", hipGetErrorString(code));
}

void usPreScanToPostScan3DConverter::GPUFillFullLookupTables()
{
  int X = m_VpreScan.getWidth();
  int Y = m_VpreScan.getHeight();
  int Z = m_VpreScan.getNumberOfFrames();

  double xmax;
  double ymin;
  double ymax;
  double zmax;

  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord(0.0, X, Z, &ymin, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z / 2.0, &ymax, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, (double)X, Z / 2.0, NULL, &xmax, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z, NULL, NULL, &zmax);

  dim3 threadsPerBlock(8, 8, 8);
  dim3 numBlocks((m_nbX+threadsPerBlock.x-1)/threadsPerBlock.x, (m_nbY+threadsPerBlock.y-1)/threadsPerBlock.y, (m_nbZ+threadsPerBlock.z-1)/threadsPerBlock.z);
  kernelPostScanVoxelFillFullLookUpTable<<<numBlocks, threadsPerBlock>>>((cudaVoxelWeightAndIndex *)m_GPULookupTables[0], (cudaVoxelWeightAndIndex *)m_GPULookupTables[1], m_nbX, m_nbY, m_nbZ, X, Y, Z, m_resolution, xmax, ymin, zmax, m_VpreScan.getFrameNumber(), m_VpreScan.getScanLineNumber(), m_VpreScan.getTransducerRadius(), m_VpreScan.getMotorRadius(), m_VpreScan.getScanLinePitch(), m_VpreScan.getAxialResolution(), m_VpreScan.getFramePitch());

  hipError_t code = hipDeviceSynchronize();
  if (code != hipSuccess) throw vpException(vpException::fatalError, "usPreScanToPostScan3DConverter::GPUFillFullLookupTables: %s", hipGetErrorString(code));
}

void usPreScanToPostScan3DConverter::GPUFullLookupTableConversion(unsigned char *dataPost, const unsigned char *dataPre)
{
  int X = m_VpreScan.getWidth();
  int Y = m_VpreScan.getHeight();
  int Z = m_VpreScan.getNumberOfFrames();

  unsigned char *dataPostDevice;
  unsigned int sizePost = m_nbX*m_nbY*m_nbZ*sizeof(unsigned char);
  unsigned char *dataPreDevice;
  unsigned int sizePre = X*Y*Z*sizeof(unsigned char);

  hipError_t codePost = hipMalloc((void **)&dataPostDevice, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: GPU post-scan memory allocation error (%d Bytes)", sizePost);
  hipError_t codePre = hipMalloc((void **)&dataPreDevice, sizePre);
  if (codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: GPU pre-scan memory allocation error (%d Bytes)", sizePre);

  codePost = hipMemset(dataPostDevice, 0, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: GPU post-scan memory set error");
  codePre = hipMemcpy(dataPreDevice, dataPre, sizePre, hipMemcpyHostToDevice);
  if (codePre != hipSuccess) {
    std::cout << codePre << std::endl;
    throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: GPU pre-scan memory copy error");
  }
  unsigned int sweepIndex = m_SweepInZdirection ? 0 : 1;
  dim3 threadsPerBlock(512);
  dim3 numBlocks((m_GPULookupTablesSize[sweepIndex]+threadsPerBlock.x-1)/threadsPerBlock.x);
  kernelPostScanVoxelFullLookUpTable<<<numBlocks, threadsPerBlock>>>(dataPostDevice, dataPreDevice, (const cudaVoxelWeightAndIndex *)(m_GPULookupTables[sweepIndex]), m_GPULookupTablesSize[sweepIndex]);

  hipError_t codeExec = hipDeviceSynchronize();
  if (codeExec != hipSuccess) throw vpException(vpException::fatalError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: %s", hipGetErrorString(codeExec));

  codePost = hipMemcpy(dataPost, dataPostDevice, sizePost, hipMemcpyDeviceToHost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUFullLookupTableConversion: GPU post-scan memory copy error");
  hipFree(dataPostDevice);
  hipFree(dataPreDevice);
}

void usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables()
{
  if (m_GPULookupTables[0] != NULL || m_GPULookupTables[1] != NULL) this->GPUFreeLookupTables();

  m_GPULookupTablesSize[0] = (long int)m_nbX * (long int)m_nbY * (long int)m_nbZ;
  m_GPULookupTablesSize[1] = m_GPULookupTablesSize[0];
  long int LUTmaxSize = m_GPULookupTablesSize[0] * sizeof(cudaVoxelWeightAndIndexReducedMemory);

  hipError_t code = hipMalloc((void **)&(m_GPULookupTables[0]), LUTmaxSize);
  if (code != hipSuccess) {
    std::cout << "Warning: usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: GPU memory allocation error for table 1 (" << LUTmaxSize << " Bytes): " << hipGetErrorString(code) << "\nwill try on RAM, performances can be impacted" << std::endl;
    code = hipMallocManaged((void **)&(m_GPULookupTables[0]), LUTmaxSize);
    if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: memory allocation error for table 1 (%d Bytes): %s", LUTmaxSize, hipGetErrorString(code));
  }

  code = hipMalloc((void **)&(m_GPULookupTables[1]), LUTmaxSize);
  if (code != hipSuccess) {
    std::cout << "Warning: usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: GPU memory allocation error for table 2 (" << LUTmaxSize << " Bytes): " << hipGetErrorString(code) << "\nwill try on RAM, performances can be impacted" << std::endl;
    code = hipMallocManaged((void **)&(m_GPULookupTables[1]), LUTmaxSize);
    if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: memory allocation error for table 2 (%d Bytes): %s", LUTmaxSize, hipGetErrorString(code));
  }

  code = hipMemset(m_GPULookupTables[0], 0, LUTmaxSize);
  if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: GPU memory set error for table 1: %s", hipGetErrorString(code));
  code = hipMemset(m_GPULookupTables[1], 0, LUTmaxSize);
  if (code != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUAllocateReducedLookupTables: GPU memory set error for table 2: %s", hipGetErrorString(code));
}

void usPreScanToPostScan3DConverter::GPUFillReducedLookupTables()
{
  int X = m_VpreScan.getWidth();
  int Y = m_VpreScan.getHeight();
  int Z = m_VpreScan.getNumberOfFrames();

  double xmax;
  double ymin;
  double ymax;
  double zmax;

  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord(0.0, X, Z, &ymin, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z / 2.0, &ymax, NULL, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, (double)X, Z / 2.0, NULL, &xmax, NULL);
  usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z, NULL, NULL, &zmax);

  dim3 threadsPerBlock(8, 8, 8);
  dim3 numBlocks((m_nbX+threadsPerBlock.x-1)/threadsPerBlock.x, (m_nbY+threadsPerBlock.y-1)/threadsPerBlock.y, (m_nbZ+threadsPerBlock.z-1)/threadsPerBlock.z);
  kernelPostScanVoxelFillReducedLookUpTable<<<numBlocks, threadsPerBlock>>>((cudaVoxelWeightAndIndexReducedMemory *)m_GPULookupTables[0], (cudaVoxelWeightAndIndexReducedMemory *)m_GPULookupTables[1], m_nbX, m_nbY, m_nbZ, X, Y, Z, m_resolution, xmax, ymin, zmax, m_VpreScan.getFrameNumber(), m_VpreScan.getScanLineNumber(), m_VpreScan.getTransducerRadius(), m_VpreScan.getMotorRadius(), m_VpreScan.getScanLinePitch(), m_VpreScan.getAxialResolution(), m_VpreScan.getFramePitch());

  hipError_t code = hipDeviceSynchronize();
  if (code != hipSuccess) throw vpException(vpException::fatalError, "usPreScanToPostScan3DConverter::GPUFillReducedLookupTables: %s", hipGetErrorString(code));
}

void usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion(unsigned char *dataPost, const unsigned char *dataPre)
{
  int X = m_VpreScan.getWidth();
  int Y = m_VpreScan.getHeight();
  int Z = m_VpreScan.getNumberOfFrames();

  unsigned char *dataPostDevice;
  unsigned int sizePost = m_nbX*m_nbY*m_nbZ*sizeof(unsigned char);
  unsigned char *dataPreDevice;
  unsigned int sizePre = X*Y*Z*sizeof(unsigned char);

  hipError_t codePost = hipMalloc((void **)&dataPostDevice, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: GPU post-scan memory allocation error (%d Bytes)", sizePost);
  hipError_t codePre = hipMalloc((void **)&dataPreDevice, sizePre);
  if (codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: GPU pre-scan memory allocation error (%d Bytes)", sizePre);

  codePost = hipMemset(dataPostDevice, 0, sizePost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: GPU post-scan memory set error");
  codePre = hipMemcpy(dataPreDevice, dataPre, sizePre, hipMemcpyHostToDevice);
  if (codePre != hipSuccess) {
    std::cout << codePre << std::endl;
    throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: GPU pre-scan memory copy error");
  }
  unsigned int sweepIndex = m_SweepInZdirection ? 0 : 1;
  dim3 threadsPerBlock(512);
  dim3 numBlocks((m_GPULookupTablesSize[sweepIndex]+threadsPerBlock.x-1)/threadsPerBlock.x);
  kernelPostScanVoxelReducedLookUpTable<<<numBlocks, threadsPerBlock>>>(dataPostDevice, dataPreDevice, (const cudaVoxelWeightAndIndexReducedMemory *)(m_GPULookupTables[sweepIndex]), m_GPULookupTablesSize[sweepIndex], X, Y);

  hipError_t codeExec = hipDeviceSynchronize();
  if (codeExec != hipSuccess) throw vpException(vpException::fatalError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: %s", hipGetErrorString(codeExec));

  codePost = hipMemcpy(dataPost, dataPostDevice, sizePost, hipMemcpyDeviceToHost);
  if (codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUReducedLookupTableConversion: GPU post-scan memory copy error");
  hipFree(dataPostDevice);
  hipFree(dataPreDevice);
}

#endif
